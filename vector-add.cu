#include <hip/hip_runtime.h>

__global__ void vectorAddKernel(float* A, float* B, float* C, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N)
        C[idx] = A[idx] + B[idx];
}

extern "C" {
void runCudaVectorAdd(float* A, float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
} 