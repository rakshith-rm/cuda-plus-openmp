#include <hip/hip_runtime.h>

__global__ void vectorAddKernel(float* A, float* B, float* C, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N)
        C[idx] = A[idx] + B[idx];
}

extern "C" {
void runCudaVectorAdd(float* A, float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void runCudaVectorAddStreams(float* A, float* B, float* C, int N, int num_streams) {
    const int chunk_size = N / num_streams;
    hipStream_t* streams = new hipStream_t[num_streams];
    float **d_A = new float*[num_streams];
    float **d_B = new float*[num_streams];
    float **d_C = new float*[num_streams];
    
    // Create streams and allocate memory for each stream
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
        hipMalloc(&d_A[i], chunk_size * sizeof(float));
        hipMalloc(&d_B[i], chunk_size * sizeof(float));
        hipMalloc(&d_C[i], chunk_size * sizeof(float));
    }
    
    // Process chunks in parallel using streams
    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        int current_chunk_size = (i == num_streams - 1) ? (N - offset) : chunk_size;
        
        // Copy data to device asynchronously
        hipMemcpyAsync(d_A[i], A + offset, current_chunk_size * sizeof(float), 
                       hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_B[i], B + offset, current_chunk_size * sizeof(float), 
                       hipMemcpyHostToDevice, streams[i]);
        
        // Launch kernel asynchronously
        int threadsPerBlock = 256;
        int blocksPerGrid = (current_chunk_size + threadsPerBlock - 1) / threadsPerBlock;
        vectorAddKernel<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(d_A[i], d_B[i], d_C[i], current_chunk_size);
        
        // Copy result back to host asynchronously
        hipMemcpyAsync(C + offset, d_C[i], current_chunk_size * sizeof(float), 
                       hipMemcpyDeviceToHost, streams[i]);
    }
    
    // Synchronize all streams
    hipDeviceSynchronize();
    
    // Cleanup
    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
        hipFree(d_A[i]);
        hipFree(d_B[i]);
        hipFree(d_C[i]);
    }
    
    delete[] streams;
    delete[] d_A;
    delete[] d_B;
    delete[] d_C;
}
} 